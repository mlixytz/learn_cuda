#include "hip/hip_runtime.h"
#include <stdio.h>
#include "error.cuh"

// (A+B)/2=C
#define N (1024 * 1024) // 每个流执行数据大小
#define FULL (N * 20)   // 全部数据大小

__global__ void kernel(int *a, int *b, int *c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
    {
        c[idx] = (a[idx] + b[idx]) / 2;
    }
}

int main()
{
    // 查询设备属性
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap)
    {
        printf("Device will not support overlap!");
        return 0;
    }
    // 初始化计时器事件
    hipEvent_t start, stop;
    float elaspsedTime;
    // 声明流和Buffer指针
    hipStream_t stream;
    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;
    // 创建计时器
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    // 初始化流
    hipStreamCreate(&stream);
    // 在GPU端申请存储
    CHECK(hipMalloc((void **)&dev_a, N * sizeof(int)));
    CHECK(hipMalloc((void **)&dev_b, N * sizeof(int)));
    CHECK(hipMalloc((void **)&dev_c, N * sizeof(int)));
    // 在CPU端申请使用锁页内存
    CHECK(hipHostAlloc((void **)&host_a, FULL * sizeof(int), hipHostMallocDefault));
    CHECK(hipHostAlloc((void **)&host_b, FULL * sizeof(int), hipHostMallocDefault));
    CHECK(hipHostAlloc((void **)&host_c, FULL * sizeof(int), hipHostMallocDefault));
    // 初始化向量A，B向量
    for (int i = 0; i < FULL; i++)
    {
        host_a[i] = rand();
        host_b[i] = rand();
    }
    // 开始计算
    hipEventRecord(start, 0);
    for (int i = 0; i < FULL; i += N)
    {
        // 将数据从CPU锁页内存中传输给GPU显存
        hipMemcpyAsync(dev_a, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(dev_b, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream);
        kernel<<<N / 256, 256, 0, stream>>>(dev_a, dev_b, dev_c);
        // 将数据从GPU显存中传输给CPU内存
        hipMemcpyAsync(host_c + i, dev_c, N * sizeof(int), hipMemcpyDeviceToHost, stream);
    }
    hipStreamSynchronize(stream);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elaspsedTime, start, stop);
    printf("Time cost: %3.1f ms\n", elaspsedTime);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipStreamDestroy(stream);

    return 0;
}