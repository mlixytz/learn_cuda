#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "error.cuh"

#define BLOCK_SIZE 16

// 每个kernel计算结果矩阵中的一个元素
// 线程数量 = 输出矩阵元素数量
__global__ void gpu_matrix_mult_shared(int *a, int *b, int *d_result, int n)
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    // 当前线程的索引
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int tmp = 0;
    int idx;

    // 把矩阵分块
    for (int sub = 0; sub < gridDim.x; ++sub)
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        tile_a[threadIdx.y][threadIdx.x] = row < n && (sub * BLOCK_SIZE + threadIdx.x) < n ? a[idx] : 0;
        idx = sub * BLOCK_SIZE + threadIdx.y * n + col;
        tile_b[threadIdx.y][threadIdx.x] = col < n && (sub * BLOCK_SIZE + threadIdx.y) < n ? b[idx] : 0;

        __syncthreads();
        for (int k = 0; k < BLOCK_SIZE; ++k)
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}

int main(int argc, char const *argv[])
{
    int m = 1000;
    int n = 1000;
    int k = 1000;

    int *h_a, *h_b, *h_c;
    CHECK(hipHostMalloc((void **)&h_a, sizeof(int) * m * n));
    CHECK(hipHostMalloc((void **)&h_b, sizeof(int) * n * k));
    CHECK(hipHostMalloc((void **)&h_c, sizeof(int) * m * k));

    hipEvent_t start, stop;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            h_a[i * n + j] = 1;
        }
    }
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            h_b[i * k + j] = 0;
        }
    }

    int *d_a, *d_b, *d_c;
    CHECK(hipMalloc((void **)&d_a, sizeof(int) * m * n));
    CHECK(hipMalloc((void **)&d_b, sizeof(int) * n * k));
    CHECK(hipMalloc((void **)&d_c, sizeof(int) * m * k));

    CHECK(hipEventRecord(start));

    CHECK(hipMemcpy(d_a, h_a, sizeof(int) * m * n, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_b, h_b, sizeof(int) * n * k, hipMemcpyHostToDevice));

    // 二维grid 二维block
    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    gpu_matrix_mult_shared<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);

    CHECK(hipMemcpy(d_c, h_c, sizeof(int) * m * k, hipMemcpyDeviceToHost));
    CHECK(hipEventRecord(stop));
    CHECK(hipEventSynchronize(stop));

    float elapsedTime;
    CHECK(hipEventElapsedTime(&elapsedTime, start, stop));
    printf("cost = %g ms.\n", elapsedTime);

    CHECK(hipFree(d_a));
    CHECK(hipFree(d_b));
    CHECK(hipFree(d_c));
    CHECK(hipHostFree(h_a));
    CHECK(hipHostFree(h_b));
    CHECK(hipHostFree(h_c));

    return 0;
}