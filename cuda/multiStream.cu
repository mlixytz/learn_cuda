#include "hip/hip_runtime.h"
#include <stdio.h>
#include "error.cuh"

// (A+B)/2=C
#define N (1024 * 1024) // 每个流执行数据大小
#define FULL (N * 20)   // 全部数据大小

__global__ void kernel(int *a, int *b, int *c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N)
    {
        c[idx] = (a[idx] + b[idx]) / 2;
    }
}

int main()
{
    // 查询设备属性
    hipDeviceProp_t prop;
    int whichDevice;
    hipGetDevice(&whichDevice);
    hipGetDeviceProperties(&prop, whichDevice);
    if (!prop.deviceOverlap)
    {
        printf("Device will not support overlap!");
        return 0;
    }
    // 初始化计时器事件
    hipEvent_t start, stop;
    float elaspsedTime;
    // 声明流和Buffer指针
    hipStream_t stream0;
    hipStream_t stream1;
    int *host_a, *host_b, *host_c;
    int *dev_a0, *dev_b0, *dev_c0;
    int *dev_a1, *dev_b1, *dev_c1;
    // 创建计时器
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));
    // 初始化流
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    // 在GPU端申请存储
    CHECK(hipMalloc((void **)&dev_a0, N * sizeof(int)));
    CHECK(hipMalloc((void **)&dev_b0, N * sizeof(int)));
    CHECK(hipMalloc((void **)&dev_c0, N * sizeof(int)));
    CHECK(hipMalloc((void **)&dev_a1, N * sizeof(int)));
    CHECK(hipMalloc((void **)&dev_b1, N * sizeof(int)));
    CHECK(hipMalloc((void **)&dev_c1, N * sizeof(int)));
    // 在CPU端申请使用锁页内存
    CHECK(hipHostAlloc((void **)&host_a, FULL * sizeof(int), hipHostMallocDefault));
    CHECK(hipHostAlloc((void **)&host_b, FULL * sizeof(int), hipHostMallocDefault));
    CHECK(hipHostAlloc((void **)&host_c, FULL * sizeof(int), hipHostMallocDefault));
    // 初始化向量A，B向量
    for (int i = 0; i < FULL; i++)
    {
        host_a[i] = rand();
        host_b[i] = rand();
    }
    // 开始计算
    hipEventRecord(start, 0);
    for (int i = 0; i < FULL; i += 2 * N)
    {
        // 将数据从CPU锁页内存中传输给GPU显存
        hipMemcpyAsync(dev_a0, host_a + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_a1, host_a + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(dev_b0, host_b + i, N * sizeof(int), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(dev_b1, host_b + i + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
        kernel<<<N / 256, 256, 0, stream0>>>(dev_a0, dev_b0, dev_c0);
        kernel<<<N / 256, 256, 0, stream1>>>(dev_a1, dev_b1, dev_c1);
        // 将数据从GPU显存中传输给CPU内存
        hipMemcpyAsync(host_c + i, dev_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(host_c + i + N, dev_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);
    }
    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elaspsedTime, start, stop);
    printf("Time cost: %3.1f ms\n", elaspsedTime);
    hipFree(dev_a0);
    hipFree(dev_b0);
    hipFree(dev_c0);
    hipFree(dev_a1);
    hipFree(dev_b1);
    hipFree(dev_c1);
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);

    return 0;
}